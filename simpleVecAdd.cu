
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>

/* cudaDeviceReset must be called before exiting in order for profilingand
   tracing tools such as Nsight and Visual Profiler to show complete traces. */
#define CUDA_DEVICE_RESET                           \
    cudaStatus = hipDeviceReset();                 \
    if (cudaStatus != hipSuccess)                  \
    {                                               \
        fprintf(stderr, "hipDeviceReset failed!"); \
        return 1;                                   \
    }

#define CUDA_CHECK_ERROR(cudaStatus)                                                         \
    if (cudaStatus != hipSuccess)                                                           \
    {                                                                                        \
        printf("%s in %s in line %d\n", hipGetErrorString(cudaStatus), __FILE__, __LINE__); \
        return 1;                                                                            \
    }

// cudaDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
#define CUDA_DEVICE_SYNCH                                                                                         \
    cudaStatus = hipDeviceSynchronize();                                                                         \
    if (cudaStatus != hipSuccess)                                                                                \
    {                                                                                                             \
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); \
        return 1;                                                                                                 \
    }

__global__ void addVec(int *C, int *A, int *B, unsigned int size);

int main()
{
    const int arraySize = 1000;
    const float block_dim = 32.0;
    hipError_t cudaStatus; // need for all cuda macors
    size_t vec_size = arraySize * sizeof(int);
    // C = A + B
    int h_A[arraySize] = {1};
    int h_B[arraySize] = {2};
    int h_C[arraySize] = {0};

    // init
    for (int i = 0; i < arraySize; i++)
    {
        h_A[i] = i * 10;
        h_B[i] = 1;
    }

    int *d_A = 0;
    int *d_B = 0;
    int *d_C = 0;
    cudaStatus = hipMalloc((void **)&d_A, vec_size);
    CUDA_CHECK_ERROR(cudaStatus)

    cudaStatus = hipMalloc((void **)&d_B, vec_size);
    CUDA_CHECK_ERROR(cudaStatus)

    cudaStatus = hipMalloc((void **)&d_C, vec_size);
    CUDA_CHECK_ERROR(cudaStatus)

    cudaStatus = hipMemcpy(d_A, h_A, vec_size, hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR(cudaStatus)
    cudaStatus = hipMemcpy(d_B, h_B, vec_size, hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR(cudaStatus)

    addVec<<<ceil(arraySize / block_dim), block_dim>>>(d_C, d_A, d_B, vec_size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    CUDA_CHECK_ERROR(cudaStatus)

    CUDA_DEVICE_SYNCH

    cudaStatus = hipMemcpy(h_C, d_C, vec_size, hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR(cudaStatus)

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    CUDA_DEVICE_RESET

    for (int i = 0; i < arraySize; i++)
    {
        printf("%d ", h_C[i]);
    }
    printf("\n");
    return 0;
}

__global__ void addVec(int *C, int *A, int *B, unsigned int size)
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
        C[index] = A[index] + B[index];
}
